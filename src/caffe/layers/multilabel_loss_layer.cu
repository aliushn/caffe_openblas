#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
void MultiLabelLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->cpu_data();
  const Dtype* target = bottom[1]->cpu_data(); 
  Dtype loss = 0;

  int dim = count/num;
/*  Dtype weight[] = {0.497, 0.3288, 0.1023, 0.0617, 0.1966, 0.1994, 0.8608, 0.8530, 0.1375, 0.1339, 
                   0.1016, 0.0692, 0.3061, 0.2962, 0.0402, 0.2375, 0.5485, 0.2957, 0.0839, 0.7494,
                   0.2759, 0.0266, 0.0765, 0.0204, 0.3633, 0.0347, 0.1418, 0.0455, 0.2161, 0.0172, 
                   0.0291, 0.5151, 0.0842, 0.4556, 0.0118};
*/
//  Dtype weight[] = {0.4865, 0.0789, 0.6699, 0.1386, 0.1123, 0.2427, 0.7945, 0.1314, 0.1352, 0.1820, 0.0539};

  int weight_size = this->layer_param_.multilabel_loss_param().weight_size();
  Dtype* weight = NULL;
  if ( weight_size > 0 )
  {
     CHECK_EQ(weight_size, dim) <<
         "weight must has the same size with channels.";
     weight = new Dtype[dim];
     for(int i = 0; i < dim; i++) 
        weight[i] = this->layer_param_.multilabel_loss_param().weight(i); 
  }
  else
  {  
     weight = new Dtype[dim];
     for(int i = 0; i < dim; i++)
        weight[i] = 0.5;
  }

  for (int i = 0; i < count; ++i) {
    if (target[i] != 0) {
    // Update the loss only if target[i] is not 0
    //  loss -= input_data[i] * ((target[i] > 0) - (input_data[i] >= 0)) -
    //      log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));

        Dtype temp = input_data[i] * ((target[i] > 0) - (input_data[i] >= 0)) -
            log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));
        if (target[i] > 0)
        {
             loss -= temp*exp( 1 - weight[i%dim]);
            // loss -= temp;
        }
        else
        {
            loss -= temp*exp(weight[i%dim]); 
            // loss -= temp;
        }
    }
  }

  if (top.size() >= 1) {
    top[0]->mutable_cpu_data()[0] = loss / num;
  }
  delete [] weight;
  // return loss / num;
}

template <typename Dtype>
void MultiLabelLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  // TODO: implement the GPU version of sigmoid_cross_entropy with -1,0,1 labels
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiLabelLossLayer);

}  // namespace caffe
